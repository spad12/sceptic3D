#include "hip/hip_runtime.h"


#include "XPlist.cuh"

__global__
void dummy_kernel2(void){;}


extern "C" void pick_gpu_(int* myid,int* mynpart)
{
	int ndevices;
	hipGetDeviceCount(&ndevices);
	int my_device = *myid;

	if(*myid < ndevices)
	{
		CUDA_SAFE_CALL(hipSetDevice(my_device));
	}
	else
	{
		*mynpart = 0;
	}
}

extern "C" void gpu_mesh_init_(long int* Mesh_ptr,
													float* phi,
													float* phiaxis,
													float* rho,
													float* rhoDiag,
													float* rmesh, // r(nrsize)
													float* rccmesh, // rcc(nrsize)
													float* thmesh, // th(nthsize)
													float* tccmesh, // tcc(nthsize)
													float* thang, // thang(nthsize)
													float* pcc, // pcc(npsisize)
													float* volinv, // volinv(nrsize)
													float* zeta, // zeta(nrsize+1)
													float* zetahalf, // zetahalf(nrsize+1)
													int* irpre, // irpre(nrpre)
													int* itpre, // itpre(ntpre)
													int* ippre, // ippre(nppre)
													float* Qcom, // Qcom(nQth)
													float* Gcom, // Gcom(nvel,nQth)
													float* Vcom, // Vcom(nvel))
													float* fparams,
													int* intparams,
													int* ierr
													)
{
	printf("sizeof ushort = %i\n",sizeof(ushort));
	int my_device;
	hipGetDevice(&my_device);
	printf("My device is %i\n",my_device);
	Mesh_data Mesh_d;
	Mesh_data* Mesh = (Mesh_data*)malloc(sizeof(Mesh_data));

	int nr,nth,npsi;
	int nrpre,ntpre,nppre;
	int nQth,nvel;

	float pi = 3.1415927;

	nr = intparams[10]+1;
	nth = intparams[11]+1;
	npsi = intparams[12]+1;

	nrpre = intparams[13];
	ntpre = intparams[14];
	nppre = intparams[15];

	nQth = intparams[17];
	nvel = intparams[16];

	Mesh_d.nr = intparams[4];
	Mesh_d.nth = intparams[5];
	Mesh_d.npsi = intparams[6];
	Mesh_d.nrused = intparams[7];
	Mesh_d.nthused = intparams[8];
	Mesh_d.npsiused = intparams[9];
	Mesh_d.nrfull = nr-1;
	Mesh_d.nthfull = nth-1;
	Mesh_d.npsifull = npsi-1;
	Mesh_d.nrpre = nrpre;
	Mesh_d.ntpre = ntpre;
	Mesh_d.nppre = nppre;
	Mesh_d.nQth = nQth;
	Mesh_d.nvel = nvel;

	Mesh_d.lat0 = intparams[18];
	Mesh_d.lap0 = intparams[19];

	Mesh_d.rfac = fparams[0];
	Mesh_d.tfac = fparams[1];
	Mesh_d.pfac = fparams[2];

	Mesh_d.debyelen = fparams[3];

	Mesh_d.bdyfc = fparams[4];
	Mesh_d.Ti = fparams[5];
	Mesh_d.vd = fparams[6];
	Mesh_d.cd = fparams[7];
	Mesh_d.cB = fparams[8];
	Mesh_d.Bz = fparams[9];

	Mesh_d.dp = 2.0*pi/Mesh_d.npsi;
	Mesh_d.dth = pi/(Mesh_d.nth-1);
	Mesh_d.dpinv = Mesh_d.npsi/2.0/pi;
	Mesh_d.dthinv = (Mesh_d.nth-1)/pi;
	Mesh_d.dpsi = pcc[2]-pcc[1];

	//printf("Mesh parameters = %i, %i, %i, %i, %i, %i, %i, %i\n",Mesh_d.nr,Mesh_d.nth,Mesh_d.npsi,nrpre,ntpre,nppre,nQth,nvel);
	//printf("dims = %i, %i, %i\n",nr,nth,npsi);


/*
	// Populate host_commons mesh variables
	Mesh_h.phi = phi;
	Mesh_h.phiaxis = phiaxis;
	Mesh_h.rho = rho;
	Mesh_h.rhoDiag = rhoDiag;
	Mesh_h.rmesh = rmesh;
	Mesh_h.rccmesh = rccmesh;
	Mesh_h.thmesh = thmesh;
	Mesh_h.tccmesh = tccmesh;
	Mesh_h.thang = thang;
	Mesh_h.pcc = pcc;
	Mesh_h.volinv = volinv;
	Mesh_h.zeta = zeta;
	Mesh_h.zetahalf = zetahalf;
	Mesh_h.irpre = irpre;
	Mesh_h.itpre = itpre;
	Mesh_h.ippre = ippre;
	Mesh_h.Qcom = Qcom;
	Mesh_h.Gcom = Gcom;
	Mesh_h.Vcom = Vcom;
*/
	/*
	 * For the GPU mesh data:
	 * Most of these are only for general storage and latter reference.
	 * Currently all of the data is wrapped in a cudaMatrix so that I can use a template structure for
	 * both the host and device data. This should make it a lot easier to make changes or additions.
	 *
	 * I think the best approach is going to be to make a new object that can store data in either a
	 * cudaMatrix or a texture bound hipArray. I need to determine whether or not we would have
	 * to rebind the hipArray to a texture reference every time step. If we do, then it probably won't
	 * be worth it to use textures.
	 */

	// Allocate gpu memory
	Mesh_d.phi.cudaMatrix_allocate(nr,nth,npsi);
	Mesh_d.phiaxis.cudaMatrix_allocate(nr,2,npsi);
	Mesh_d.rho.cudaMatrix_allocate(nr,nth,npsi);
	Mesh_d.rhoDiag.cudaMatrix_allocate(nr,nth,npsi);
	Mesh_d.rmesh.cudaMatrix_allocate(nr,1,1);
	Mesh_d.rccmesh.cudaMatrix_allocate(nr,1,1);
	Mesh_d.thmesh.cudaMatrix_allocate(nth,1,1);
	Mesh_d.tccmesh.cudaMatrix_allocate(nth,1,1);
	Mesh_d.thang.cudaMatrix_allocate(nth,1,1);
	Mesh_d.pcc.cudaMatrix_allocate(npsi,1,1);
	Mesh_d.volinv.cudaMatrix_allocate(nr,1,1);
	Mesh_d.zeta.cudaMatrix_allocate(nr+1,1,1);
	Mesh_d.zetahalf.cudaMatrix_allocate(nr+1,1,1);
	Mesh_d.irpre.cudaMatrix_allocate(nrpre,1,1);
	Mesh_d.itpre.cudaMatrix_allocate(ntpre,1,1);
	Mesh_d.ippre.cudaMatrix_allocate(nppre,1,1);
	Mesh_d.Qcom.cudaMatrix_allocate(nQth,1,1);
	Mesh_d.Gcom.cudaMatrix_allocate(nvel,nQth,1);
	Mesh_d.Vcom.cudaMatrix_allocate(nvel,1,1);

	Mesh_d.psum.cudaMatrix_allocate(nr-2,nth-2,npsi-2);

	Mesh_d.nincell.cudaMatrix_allocate(nth-1,npsi-1,1);
	Mesh_d.vrincell.cudaMatrix_allocate(nth-1,npsi-1,1);
	Mesh_d.vr2incell.cudaMatrix_allocate(nth-1,npsi-1,1);

	Mesh_d.nbins = ((Mesh_d.nr)*(Mesh_d.nth)*(Mesh_d.npsi)+8*8*8-1)/(8*8*8);
	CUDA_SAFE_CALL(hipMalloc((void**)&(Mesh_d.bins),(Mesh_d.nbins+8*8*8)*sizeof(Particlebin)));

	// Copy Mesh data to the GPU
	Mesh_d.rmesh.cudaMatrixcpy(rmesh,hipMemcpyHostToDevice);
	Mesh_d.rccmesh.cudaMatrixcpy(rccmesh,hipMemcpyHostToDevice);
	Mesh_d.thmesh.cudaMatrixcpy(thmesh,hipMemcpyHostToDevice);
	Mesh_d.tccmesh.cudaMatrixcpy(tccmesh,hipMemcpyHostToDevice);
	Mesh_d.thang.cudaMatrixcpy(thang,hipMemcpyHostToDevice);
	Mesh_d.pcc.cudaMatrixcpy(pcc,hipMemcpyHostToDevice);
	Mesh_d.volinv.cudaMatrixcpy(volinv,hipMemcpyHostToDevice);
	Mesh_d.zeta.cudaMatrixcpy(zeta,hipMemcpyHostToDevice);
	Mesh_d.zetahalf.cudaMatrixcpy(zetahalf,hipMemcpyHostToDevice);
	Mesh_d.irpre.cudaMatrixcpy(irpre,hipMemcpyHostToDevice);
	Mesh_d.itpre.cudaMatrixcpy(itpre,hipMemcpyHostToDevice);
	Mesh_d.ippre.cudaMatrixcpy(ippre,hipMemcpyHostToDevice);
	Mesh_d.Qcom.cudaMatrixcpy(Qcom,hipMemcpyHostToDevice);
	Mesh_d.Gcom.cudaMatrixcpy(Gcom,hipMemcpyHostToDevice);
	Mesh_d.Vcom.cudaMatrixcpy(Vcom,hipMemcpyHostToDevice);




	// Store the pointer to the mesh data and wrap it as an int
	*Mesh = Mesh_d;

	*Mesh_ptr = (long int)Mesh;


}


extern "C" void gpu_particle_list_init_(long int* particles_out,int* nptcls)
{

	printf("Setting up particle list\n");
	XPlist* particles = (XPlist*)malloc(sizeof(XPlist));

	particles->allocate(*nptcls);

	*particles_out = (long int)particles;
	printf("Finished setting up particle list\n");

	size_t free = 0;
	size_t total = 0;
	// See how much memory is allocated / free
	hipMemGetInfo(&free,&total);
	printf("Free Memory = %i mb\nUsed mememory = %i mb\n",(int)(free)/(1<<20),(int)(total-free)/(1<<20));

	printf("Setting up Mesh Arrays\n");

}

extern "C" void gpu_Diagnostics_init()
{

}

extern "C" void gpu_Distribution_fnct_init()
{

}















