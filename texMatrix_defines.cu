#include "hip/hip_runtime.h"
#include "texMatrix.cuh"


// We need a dummy kernal to force ptxas to actually compile this
__global__
void dummy_kernel(void){;}


int next_tex1D = 0;
int next_tex2D = 0;
int next_tex1DLayered = 0;
int next_tex2DLayered = 0;
int next_tex3D = 0;

__inline__ __host__
int get_next_tex(int textureType)
{
	int result;
	switch(textureType)
	{
	case hipTextureType1D:
		result = next_tex1D;
		next_tex1D++;
		break;
	case hipTextureType1DLayered:
		result = next_tex1DLayered;
		next_tex1DLayered++;
		break;
	case hipTextureType2D:
		result = next_tex2D;
		next_tex2D++;
		break;
	case hipTextureType2DLayered:
		result = next_tex2DLayered;
		next_tex2DLayered++;
		break;
	case hipTextureType3D:
		result = next_tex3D;
		next_tex3D++;
		break;
	default:
		break;
	}

	return result;
}
__host__
void texMatrix::get_tex_string(char* texrefstring,char* texfetchstring)
{
	switch(textureType)
	{
	case hipTextureType1D:
		sprintf(texrefstring,"texref1D%i",texture_ref_index);
		sprintf(texfetchstring,"fetchtexref1DPtr%i",texture_ref_index);
		break;
	case hipTextureType1DLayered:
		sprintf(texrefstring,"texref1DLayered%i",texture_ref_index);
		sprintf(texfetchstring,"fetchtexref1DLayeredPtr%i",texture_ref_index);
		break;
	case hipTextureType2D:
		sprintf(texrefstring,"texref2D%i",texture_ref_index);
		sprintf(texfetchstring,"fetchtexref2DPtr%i",texture_ref_index);
		break;
	case hipTextureType2DLayered:
		sprintf(texrefstring,"texref2DLayered%i",texture_ref_index);
		sprintf(texfetchstring,"fetchtexref2DLayeredPtr%i",texture_ref_index);
		break;
	case hipTextureType3D:
		sprintf(texrefstring,"texref3D%i",texture_ref_index);
		sprintf(texfetchstring,"fetchtexref3DPtr%i",texture_ref_index);
		break;
	default:
		break;
	}

}


__host__
void texMatrix::allocate(int nx,int ny,int nz,int textype)
{
	hipExtent extent;

	textureType = textype;

	dims.x = nx;
	dims.y = ny;
	dims.z = nz;

	char* texrefstring = (char*)malloc(sizeof(char)*30);
	char* texfetchstring = (char*)malloc(sizeof(char)*30);

	texture_ref_index = get_next_tex(textureType);

	get_tex_string(texrefstring,texfetchstring);

	char* symbol = texrefstring;

	//CUDA_SAFE_CALL(hipGetSymbolAddress((void**)&fetchFunction,texfetchstring));
	CUDA_SAFE_CALL(hipMemcpyFromSymbol((void*)&fetchFunction,texfetchstring,sizeof(texFunctionPtr)));
	//printf(" fill2D nx = %i, ny = %i \n", nx,ny);

	switch(textureType)
	{
	case hipTextureType1D:
		extent = make_hipExtent(nx,0,0);
		break;
	case hipTextureType1DLayered:
		extent = make_hipExtent(nx,0,ny);
		break;
	case hipTextureType2D:
		extent = make_hipExtent(nx,ny,0);
		break;
	case hipTextureType2DLayered:
		extent = make_hipExtent(nx,ny,nz);
		break;
	case hipTextureType3D:
		extent = make_hipExtent(nx,ny,nz);
		break;
	default:
		break;
	}

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();


	CUDA_SAFE_CALL(hipMalloc3DArray(&cuArray,&desc,extent));

	is_bound = 0;

	free(texrefstring);
	free(texfetchstring);
}


__host__
void texMatrix::copy(float* src,enum hipMemcpyKind kind)
{
	int nx = dims.x;
	int ny = dims.y;
	int nz = dims.z;

	hipMemcpy3DParms params = {0};
	const textureReference* texRefPtr;
	hipChannelFormatDesc channelDesc;
	CUDA_SAFE_CALL(hipGetChannelDesc(&channelDesc, cuArray));

	char* texrefstring = (char*)malloc(sizeof(char)*30);
	char* texfetchstring = (char*)malloc(sizeof(char)*30);

	// Get the symbol for the texture reference
	get_tex_string(texrefstring,texfetchstring);

	char* symbol = texrefstring;

	// Get the texture reference
	CUDA_SAFE_CALL(hipGetTextureReference(&texRefPtr, HIP_SYMBOL(symbol)));

	// Make sure that the texture isn't already bound
	if(is_bound)
		CUDA_SAFE_CALL(hipUnbindTexture(texRefPtr));

	params.dstArray = cuArray;
	params.srcPtr.ptr = (void**)src;
	params.srcPtr.pitch = nx*sizeof(float);
	params.srcPtr.xsize = nx;
	params.kind = kind;

	switch(textureType)
	{
	case hipTextureType1D:
		params.srcPtr.ysize = 1;
		params.extent = make_hipExtent(nx,1,1);
		break;
	case hipTextureType1DLayered:
		params.srcPtr.ysize = 1;
		params.extent = make_hipExtent(nx,1,ny);
		break;
	case hipTextureType2D:
		params.srcPtr.ysize = ny;
		params.extent = make_hipExtent(nx,ny,1);
		break;
	case hipTextureType2DLayered:
		params.srcPtr.ysize = ny;
		//params.extent = make_hipExtent(nx,ny,nz);
		break;
	case hipTextureType3D:
		params.srcPtr.ysize = ny;
		params.extent = make_hipExtent(nx,ny,nz);
		break;
	default:
		break;
	}

	printf("ref # %i\n",texture_ref_index);

	// Do the copy
	CUDA_SAFE_CALL(hipMemcpy3D(&params));

	hipDeviceSynchronize();

	// Get the hipArray's channel descriptor
	CUDA_SAFE_CALL(hipGetChannelDesc(&channelDesc, cuArray));

	// Bind the hipArray to the texture reference
	CUDA_SAFE_CALL(hipBindTextureToArray(texRefPtr, cuArray, &channelDesc));
	hipDeviceSynchronize();

	is_bound = 1;

	free(texrefstring);
	free(texfetchstring);

}






